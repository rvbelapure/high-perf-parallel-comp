#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

#define TILE_DIM   32	// make side of matrix multiple of 32
#define BLOCK_ROWS 8

#define DEBUG 0


typedef float dtype;

__global__
void matTrans(dtype* AT, dtype* A, int N)  {
	int horloc = blockIdx.x * TILE_DIM + threadIdx.x;
	int verloc = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
		AT[horloc * width + (verloc + j)] = A[(verloc + j)*width + horloc];
}

__global__
void warmup() {
	for(int i = 0 ; i < 1000 ; i++);
}

void printArray(int N, dtype *A, char *message)
{
	printf("%s : \n", message);
	for(int i = 0; i < N ; i++)
	{
		for(int j = 0 ; j < N ; j++)
			printf("%ld ",A[i * N + j]);
		printf("\n");
	}
	printf("---------------------------\n");
}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}


void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}



void
gpuTranspose (dtype* A, dtype* AT, int N)
{
  struct stopwatch_t* timer = NULL;
  long double t_gpu;

  void (*kernel)(dtype* , dtype* , int );		// kernel pointer - change the association to determine which kernel to launch
  kernel = &matTrans;

  #if DEBUG
  printArray(N, A, "input");
  #endif

  /* Now we have A as input array and AT as the output array on host side. 
     N is the length of side for square matrix N * N */
  /* 0. As per our algorithm, we have to make sure that side of the matrix is multiple of our TILE_DIM.
     Thus, we pad the matrix with extra elements */
  int tiled_size;
  if (N % TILE_DIM == 0)
	  tiled_size = N;
  else
  {
	  tiled_size = ((N / TILE_DIM) + 1) * TILE_DIM;
  }
  dtype * padded_input = (dtype *) malloc( tiled_size * tiled_size * sizeof(dtype));
  // we can not use memcpy as we should not copy into padded region
  for(int i = 0 ; i < N ; i++)
	  for(int j = 0 ; j < N ; j++)
		  padded_input[i * tiled_size + j] = A[i * N + j];	
  #if DEBUG
  printArray(tiled_size, padded_input, "padded input");
  #endif


  /* 1. allocate device input output arrays */
  dtype *d_A, *d_AT;
  CUDA_CHECK_ERROR(hipMalloc((void **) &d_A, tiled_size * tiled_size * sizeof(dtype)));
  CUDA_CHECK_ERROR(hipMalloc((void **) &d_AT, tiled_size * tiled_size * sizeof(dtype)));

  /* 2. Fill the device input array */
  CUDA_CHECK_ERROR(hipMemcpy(d_A, padded_input, tiled_size * tiled_size * sizeof(dtype), hipMemcpyHostToDevice));

  /* 3. Calculate gridDim and blockDim here */
  dim3 grdDim( tiled_size / TILE_DIM, tiled_size / TILE_DIM, 1);
  dim3 blkDim( TILE_DIM, BLOCK_ROWS, 1);
	
  /* 4. Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

  /* 5. warmup */
  hipFree(0);
  warmup<<<25,25>>>();
  hipDeviceSynchronize ();

  stopwatch_start (timer);
  /* 6. run your kernel here */
  kernel<<<grdDim, blkDim>>>(d_AT, d_A, tiled_size);
  hipDeviceSynchronize ();
  t_gpu = stopwatch_stop (timer);
  fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );

  /* 7. copy the answer back to host array for further checking */
  CUDA_CHECK_ERROR( hipMemcpy( padded_input, d_AT, tiled_size * tiled_size * sizeof(dtype), hipMemcpyDeviceToHost));
  for(int i = 0 ; i < N ; i++)
	  for(int j = 0 ; j < N ; j++)
		  AT[i * N + j] = padded_input[i * tiled_size + j];
  #if DEBUG
  printArray(tiled_size, padded_input, "padded output");
  printArray(N, AT, "output");
  #endif

  /* 8. Free the device memory */
  free(padded_input);
  CUDA_CHECK_ERROR( hipFree(d_A));
  CUDA_CHECK_ERROR( hipFree(d_AT));
}

int 
main(int argc, char** argv)
{
  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;


	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
