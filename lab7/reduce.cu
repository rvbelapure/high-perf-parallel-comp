#include "hip/hip_runtime.h"
#include "driver.h"
#include "reduce.h"
#include "cuda_utils.h"

dtype
reduceCpu (dtype* h_A, unsigned int N)
{
  int i;
  dtype ans;

  ans = (dtype) 0.0;
  for(i = 0; i < N; i++) {
    ans += h_A[i];
  }

  return ans;
}

__global__ void 
reduceNaiveKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride;
	

	/* load data to buffer */
	if(tid < N) {
		buffer[threadIdx.x] = In[tid];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();

	/* reduce in shared memory */
	for(stride = 1; stride < blockDim.x; stride *= 2) {
		if(threadIdx.x % (stride * 2) == 0) {
			buffer[threadIdx.x] += buffer[threadIdx.x + stride];
		}
		__syncthreads ();
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}

dtype
reduceNaive (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = N;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	reduceNaiveKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNaiveKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNaiveKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNaiveKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNaiveKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceNonDivergeKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride;
	unsigned int index;
	

	/* load data to buffer */
	if(tid < N) {
		buffer[threadIdx.x] = In[tid];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();

	/* reduce in shared memory */
	for(stride = 1; stride < blockDim.x; stride *= 2) {
		index = threadIdx.x * 2 * stride;
		if(index < blockDim.x) {
			buffer[index] += buffer[index + stride];
		}
		__syncthreads ();
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceNonDiverge (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = N;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	reduceNonDivergeKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNonDivergeKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNonDivergeKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNonDivergeKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceNonDivergeKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceSeqAddKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int stride;
	

	/* load data to buffer */
	if(tid < N) {
		buffer[threadIdx.x] = In[tid];
	} else {
		buffer[threadIdx.x] = (dtype) 0.0;
	}
	__syncthreads ();

	/* reduce in shared memory */
	for(stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if(threadIdx.x < stride) {
			buffer[threadIdx.x] += buffer[threadIdx.x + stride];
		}
		__syncthreads ();
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceSeqAdd (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = N;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	reduceSeqAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceSeqAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceSeqAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceSeqAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceSeqAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceFirstAddKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	unsigned int stride;
	dtype tmp;
	

	/* load data to buffer */
	tmp = In[tid];	
	if(tid + blockDim.x < N) {
		tmp += In[tid + blockDim.x];
	}
	buffer[threadIdx.x] = tmp;
	__syncthreads ();

	/* reduce in shared memory */
	for(stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if(threadIdx.x < stride) {
			buffer[threadIdx.x] += buffer[threadIdx.x + stride];
		}
		__syncthreads ();
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceFirstAdd (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = (N + 1) / 2;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	reduceFirstAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceFirstAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceFirstAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceFirstAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceFirstAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}

__global__ void 
reduceUnrollLastKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	unsigned int stride;
	dtype tmp;
	

	/* load data to buffer */
	tmp = In[tid];	
	if(tid + blockDim.x < N) {
		tmp += In[tid + blockDim.x];
	}
	buffer[threadIdx.x] = tmp;
	__syncthreads ();

	/* reduce in shared memory */
	for(stride = blockDim.x / 2; stride > 32; stride >>= 1) {
		if(threadIdx.x < stride) {
			buffer[threadIdx.x] += buffer[threadIdx.x + stride];
		}
		__syncthreads ();
	}
	/* warp is unrolled */
	if(threadIdx.x < 32) {
		volatile dtype *sm = buffer;
		sm[threadIdx.x] += sm[threadIdx.x + 32];
		sm[threadIdx.x] += sm[threadIdx.x + 16];
		sm[threadIdx.x] += sm[threadIdx.x + 8];
		sm[threadIdx.x] += sm[threadIdx.x + 4];
		sm[threadIdx.x] += sm[threadIdx.x + 2];
		sm[threadIdx.x] += sm[threadIdx.x + 1];
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceUnrollLast (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = (N + 1) / 2;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	reduceUnrollLastKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollLastKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollLastKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollLastKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollLastKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}


__global__ void 
reduceUnrollAllKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	dtype tmp;
	

	/* load data to buffer */
	tmp = In[tid];	
	if(tid + blockDim.x < N) {
		tmp += In[tid + blockDim.x];
	}
	buffer[threadIdx.x] = tmp;
	__syncthreads ();

	/* reduce in shared memory */
	if(BS >= 1024) {
		if(threadIdx.x < 512) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 512];
			__syncthreads ();
		}
	}	
	if(BS >= 512) {
		if(threadIdx.x < 256) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 256];
			__syncthreads ();
		}
	}	
	if(BS >= 256) {
		if(threadIdx.x < 128) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 128];
			__syncthreads ();
		}
	}	
	if(BS >= 128) {
		if(threadIdx.x < 64) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 64];
			__syncthreads ();
		}
	}	

	/* warp is unrolled */
	if(threadIdx.x < 32) {
		volatile dtype *sm = buffer;
		sm[threadIdx.x] += sm[threadIdx.x + 32];
		sm[threadIdx.x] += sm[threadIdx.x + 16];
		sm[threadIdx.x] += sm[threadIdx.x + 8];
		sm[threadIdx.x] += sm[threadIdx.x + 4];
		sm[threadIdx.x] += sm[threadIdx.x + 2];
		sm[threadIdx.x] += sm[threadIdx.x + 1];
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceUnrollAll (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int nThreads, tbSize, nBlocks;
	dtype ans;


	nThreads = (N + 1) / 2;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	reduceUnrollAllKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollAllKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollAllKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollAllKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceUnrollAllKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}



__global__ void 
reduceMultAddKernel (dtype* In, dtype *Out, unsigned int N)
{
	__shared__ dtype buffer[BS];
	unsigned int tid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	dtype tmp;

	/* load data to buffer */
	tmp = 0.0;
	while(tid < N) {
		tmp += In[tid];
		if((tid + blockDim.x) < N) {
			tmp += In[tid + blockDim.x];
		}
		tid += blockDim.x * 2 * gridDim.x;
	}
	buffer[threadIdx.x] = tmp;
	__syncthreads ();

	/* reduce in shared memory */
	if(BS >= 1024) {
		if(threadIdx.x < 512) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 512];
			__syncthreads ();
		}
	}	
	if(BS >= 512) {
		if(threadIdx.x < 256) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 256];
			__syncthreads ();
		}
	}	
	if(BS >= 256) {
		if(threadIdx.x < 128) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 128];
			__syncthreads ();
		}
	}	
	if(BS >= 128) {
		if(threadIdx.x < 64) {
			buffer[threadIdx.x] += buffer[threadIdx.x + 64];
			__syncthreads ();
		}
	}	

	/* warp is unrolled */
	if(threadIdx.x < 32) {
		volatile dtype *sm = buffer;
		sm[threadIdx.x] += sm[threadIdx.x + 32];
		sm[threadIdx.x] += sm[threadIdx.x + 16];
		sm[threadIdx.x] += sm[threadIdx.x + 8];
		sm[threadIdx.x] += sm[threadIdx.x + 4];
		sm[threadIdx.x] += sm[threadIdx.x + 2];
		sm[threadIdx.x] += sm[threadIdx.x + 1];
	}

	/* store back the reduced result */
	if(threadIdx.x == 0) {
		Out[blockIdx.x] = buffer[0];
	}
}



dtype
reduceMultAdd (dtype* d_In, dtype* d_Out, dtype* h_Out, unsigned int N)
{
	unsigned int nThreads, tbSize, nBlocks;
	dtype ans;


	// nThreads = (N + 1) / 32;
	nThreads = 16384;
	tbSize = BS;
	nBlocks = (nThreads + tbSize - 1) / tbSize;

	dim3 grid (nBlocks);
	dim3 block (tbSize);

	reduceMultAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceMultAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceMultAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceMultAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();
	reduceMultAddKernel <<<grid, block>>> (d_In, d_Out, N);
	hipDeviceSynchronize ();

	CUDA_CHECK_ERROR (hipMemcpy (h_Out, d_Out, nBlocks * sizeof (dtype),
																hipMemcpyDeviceToHost));

	ans = reduceCpu (h_Out, nBlocks);

	return ans;

}





void
initCudaArray (dtype **d_A, dtype *h_A, unsigned int N)
{
	CUDA_CHECK_ERROR (hipMalloc ((void**) d_A, N * sizeof (dtype)));
	CUDA_CHECK_ERROR (hipMemcpy (*d_A, h_A, N * sizeof (dtype),
																hipMemcpyHostToDevice));
}

void
cudaReduction (dtype *A, unsigned int N, unsigned int OPT, dtype *ret)
{
	dtype *h_Out, *d_Out;
	unsigned int nBlocks;

	hipEvent_t start, stop;
	float elapsedTime;

	dtype ans;

	nBlocks = (N + BS - 1) / BS;
	h_Out = (dtype*) malloc (nBlocks * sizeof (dtype));
	CUDA_CHECK_ERROR (hipMalloc ((void**) &d_Out, nBlocks * sizeof (dtype)));
	
	CUDA_CHECK_ERROR (hipEventCreate (&start));
	CUDA_CHECK_ERROR (hipEventCreate (&stop));

	fprintf (stderr, "Executing test case [%d]\n", OPT);
	fprintf (stderr, "[1]: Naive | [2]: Non-divergent | [3]: Sequential Add. | [4]: First add | [5]: Unroll last warp | [6]: Complete unroll | [7] Multiple Adds\n");

	
	CUDA_CHECK_ERROR (hipEventRecord (start, 0));
	/* execute kernel */
	switch (OPT) {
		case 1:
			ans = reduceNaive (A, d_Out, h_Out, N);	
			break;
		case 2:
			ans = reduceNonDiverge (A, d_Out, h_Out, N);	
			break;
		case 3:
			ans = reduceSeqAdd (A, d_Out, h_Out, N);	
			break;
		case 4:
			ans = reduceFirstAdd (A, d_Out, h_Out, N);	
			break;
		case 5:
			ans = reduceUnrollLast (A, d_Out, h_Out, N);	
			break;
		case 6:
			ans = reduceUnrollAll (A, d_Out, h_Out, N);	
			break;
		case 7:
			ans = reduceMultAdd (A, d_Out, h_Out, N);	
			break;
		default:
			ans = reduceNaive (A, d_Out, h_Out, N);	
	} 
	CUDA_CHECK_ERROR (hipEventRecord (stop, 0));
	CUDA_CHECK_ERROR (hipEventSynchronize (stop));
	CUDA_CHECK_ERROR (hipEventElapsedTime (&elapsedTime, start, stop));
	elapsedTime = elapsedTime / 5;


	fprintf (stderr, "Execution time: %f ms\n", elapsedTime);
	fprintf (stderr, "Equivalent performance: %f GB/s\n", 
						(N * sizeof (dtype) / elapsedTime) * 1e-6);

	CUDA_CHECK_ERROR (hipEventDestroy (start));
	CUDA_CHECK_ERROR (hipEventDestroy (stop));

	free (h_Out);
	CUDA_CHECK_ERROR (hipFree (d_Out));

	*ret = ans;	
}


